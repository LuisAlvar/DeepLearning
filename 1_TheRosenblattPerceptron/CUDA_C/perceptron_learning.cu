/*
nvcc perceptron_learning.cu -o ./bin/perceptron_learning.exe
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N_INPUT_PERCEPTRON 2
const double LEARNING_RATE = 0.1;

/// @brief Implementation of Perceptron Function
/// @param w Is the weight vector
/// @param x Is the input vector
/// @return Return -1 or 1
double compute_output(double* w, double* x, int size)
{
  double z = 0.0;
  double* w_arry_ptr = w;
  double* x_arry_ptr = x;
  for (int index = 0; index < size; ++index)
  {
    z += (*w_arry_ptr) * (*x_arry_ptr); // Compute sum of weighted inputs
    ++w_arry_ptr;
    ++x_arry_ptr;
  }
  // Apply sign function
  if (z < 0)
  {
    return -1.0;
  }
  else {
    return 1.0;
  }
}

/// @brief Shows the current values of the weight array
/// @param w A point to an array of doubles - the weight array
/// @param size The size of the array of doubles
void showLearning(double* w, int size)
{
  for (int i = 0; i < size; ++i)
  {
    printf("w%d = %.2f |", i, w[i]);
    fflush(stdout);
  }
  printf("\n");
}

/// @brief Swaps the two given integers using XOR method
/// @param a Any given integer
/// @param b Any given integer
void Swap(int* a, int* b)
{
  (*a)^=(*b);
  (*b)^=(*a);
  (*a)^=(*b);
}

/// @brief Shuffle the given array of integers
/// @param array A pointer to an array of integers
/// @param size The size of the given array
void arrayShuffle(int* array, int size)
{
  srand(time(NULL));
  for (int index = size - 1; index > 0; --index)
  {
    int rand_within_range_j = rand() % index;
    Swap(&array[index], &array[rand_within_range_j]);
  }
}

// This is just to implement the perceptron function in CUDA C
// ONLY used the CPU. If w
int main(void)
{

  double weights[N_INPUT_PERCEPTRON+1] = {0.2, -0.6, 0.25}; // randomized weight vector
  double x_train[4][N_INPUT_PERCEPTRON+1] = {
    {1.0,-1.0,-1.0},
    {1.0,-1.0,1.0},
    {1.0,1.0,-1.0},
    {1.0,1.0,1.0},
  };
  double y_train[4] = {1.0,1.0,1.0,-1.0};

  double* w_vector_ptr, *input_vector_ptr, *output_vector_ptr;

  w_vector_ptr = weights;
  output_vector_ptr = y_train;
  showLearning(w_vector_ptr, N_INPUT_PERCEPTRON+1);

  // Perceptron Learning Algo
  int index_list[4] = {0, 1, 2, 3};
  bool all_correct = false;
  while (!all_correct)
  {
    all_correct = true;
    arrayShuffle(index_list, sizeof(index_list)/sizeof(int));
    for (size_t i = 0; i < sizeof(index_list)/sizeof(int); ++i)
    {
      input_vector_ptr = x_train[i];
      double y = y_train[i];
      double p_output = compute_output(w_vector_ptr, input_vector_ptr, N_INPUT_PERCEPTRON + 1);
      if (p_output != y)
      {
        for (size_t i = 0; i < sizeof(weights)/sizeof(double); ++i)
        {
          weights[i] += (y * LEARNING_RATE * (*input_vector_ptr));
          ++input_vector_ptr;
        }
        all_correct = false;
        showLearning(w_vector_ptr, N_INPUT_PERCEPTRON + 1);
      }
    } 
  }
  
  return 0;
}