/*
nvcc perceptron.cu -o ./bin/perceptron.exe
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

static void ErrorHandler(hipError_t err, const char* file, int line)
{
  if (err != hipSuccess)
  {
    printf("file(%s) at line %d: %s", file, line, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
#define ERROR_HANDLER( err ) (ErrorHandler(err, __FILE__, __LINE__))
#define N_INPUT 3

__global__ void DotProduct(double*a, double*b, double*c)
{
  int id = blockIdx.x;
  if (id < N_INPUT)
  {
    c[id] = a[id] * b[id];
  }
}

// Not the best utilization of having the GPU perform a dot product of this size.
// This is just to implement the perceptron function in CUDA C
int main(void)
{
  double* weight_vector;
  double* input_vector;
  double* result_vector;

  double* dev_w, *dev_x, *dev_result;

  // allocate memory on the CPU side 
  weight_vector = (double*)malloc(N_INPUT * sizeof(double));
  input_vector = (double*)malloc(N_INPUT * sizeof(double));
  result_vector = (double*)malloc(N_INPUT * sizeof(double));

  weight_vector[0] = 0.9;
  weight_vector[1] = -0.6;
  weight_vector[2] = -0.5;

  input_vector[0] = 1.0;
  input_vector[1] = 1.0;
  input_vector[2] = 1.0;

  result_vector[0] = 0.0;
  result_vector[1] = 0.0;
  result_vector[2] = 0.0;


  // allocate the memory on the GPU
  ERROR_HANDLER(hipMalloc((void**)&dev_w, N_INPUT*sizeof(double)));
  ERROR_HANDLER(hipMalloc((void**)&dev_x, N_INPUT*sizeof(double)));
  ERROR_HANDLER(hipMalloc((void**)&dev_result, N_INPUT*sizeof(double)));

  // copy data from CPU to GPU
  ERROR_HANDLER(hipMemcpy(dev_w, weight_vector, N_INPUT * sizeof(double), hipMemcpyHostToDevice));
  ERROR_HANDLER(hipMemcpy(dev_x, input_vector, N_INPUT * sizeof(double), hipMemcpyHostToDevice));

  DotProduct<<<N_INPUT,1>>>(dev_w, dev_x, dev_result);

  ERROR_HANDLER(hipMemcpy(result_vector, dev_result, N_INPUT * sizeof(double), hipMemcpyDeviceToHost));

  double z = 0.0;

  for (int i = 0; i < N_INPUT; i++)
  {
    printf("%.2f| ", result_vector[i]);
    z += result_vector[i];
  }
  printf("z: %.2f\n", z);

  if (z < 0)
  {
    printf("%d\n", -1);
  }
  else 
  {
    printf("%d\n", 1);
  }
  
  return 0;
}