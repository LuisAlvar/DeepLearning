#include "hip/hip_runtime.h"
/*
nvcc perceptron.cu -o ./bin/perceptron.exe
*/
#include <stdio.h>

static void ErrorHandler(hipError_t err, const char* file, int line)
{
  if (err != hipSuccess)
  {
    printf("file(%s) at line %d: %s", file, line, hipGetErrorString(err));
    exit(EXIST_FAILURE);
  }
}
#define ERROR_HANDLER( err ) (ErrorHandler(err, __FILE__, __LINE__))

#define N_INPUT 3

__global__ void DotProduct(int*a, int*b, int*c)
{
  int id = blockId.x;
  if (id < N_INPUT)
  {
    c[id] = a[id] * b[id];
  }
}

int main(void)
{
  double* weight_vector;
  double* input_vector;
  double* result_vector;

  double* dev_w, *dev_x, *dev_result;

  // allocate memory on the CPU side 
  weight_vector = (double*)malloc(N_INPUT * sizeof(double));
  input_vector = (double*)malloc(N_INPUT * sizeof(double));
  result_vector = (double*)malloc(N_INPUT * sizeof(double));

  weight_vector[0] = 0.9;
  weight_vector[1] = -0.6;
  weight_vector[2] = -0.5;

  input_vector[0] = 1.0;
  input_vector[1] = 1.0;
  input_vector[2] = 1.0;

  result_vector[0] = 0.0;
  result_vector[1] = 0.0;
  result_vector[2] = 0.0;


  // allocate the memory on the GPU
  ERROR_HANDLER(hipMalloc((void**)&dev_w, N_INPUT*sizeof(double)));
  ERROR_HANDLER(hipMalloc((void**)&dev_w, N_INPUT*sizeof(double)));
  ERROR_HANDLER(hipMalloc((void**)&dev_result, N_INPUT*sizeof(double)));

  // copy data from CPU to GPU
  ERROR_HANDLER(hipMemcpy(dev_w, weight_vector, N * sizeof(double), hipMemcpyHostToDevice));
  ERROR_HANDLER(hipMemcpy(dev_x, input_vector, N * sizeof(double), hipMemcpyHostToDevice));

  DotProduct<<<N_INPUT,1>>>(dev_w, dev_x, dev_result);

  ERROR_HANDLER(hipMemcpy(result_vector, dev_result, N * sizeof(double), hipMemcpyDeviceToHost));

  double z = 0.0;

  for (int i = 0; i < N_INPUT; i++)
  {
    z += result_vector[i];
  }

  if (z < 0)
  {
    printf("%d", -1);
  }
  else 
  {
    printf("%d", 1);
  }
  
  return 0;
}